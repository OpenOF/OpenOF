#include "hip/hip_runtime.h"
/*
 *   OpenOF - Open Optimization Framework
 *   Copyright (C) 2012 C. Wefelscheid
 *
 *   This file is part of OpenOF.
 *

 */




#include <cusp/coo_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include <cusp/io/matrix_market.h>

#include <cusp/precond/ainv.h>
#include <cusp/precond/diagonal.h>
#include <cusp/blas.h>
#include <cusp/print.h>
#include <cusp/krylov/cg.h>

#include <thrust/system_error.h>
#include <thrust/transform.h>
#include <thrust/generate.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/for_each.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>


#include "Levmar.h"
#include "meas_func_src.h"
#include "structs_func_src.h"
#include "general_functors.h"

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

Levmar::Levmar() {
	kmax=50;
	v=2;
	eps1=1e-10;
	eps2=1e-10;
	tau=0.001;

	cg_it=150;
	cg_thresh_rel=1e-12;
	cg_thresh_abs=1e-12;
	

	verbose=0;
	nrParaOpt=0;
	nrParaJac=0;
	nrParaFunc=0;

}

void createMatrix(memory_vector<int> &d_jac_row_ind,
		memory_vector<int> &d_jac_col_ind,
		memory_vector<oof_float> &d_jac_value,cusp::coo_matrix<int, oof_float, cusp_memory> &jac){

	thrust::copy(d_jac_row_ind.begin(),d_jac_row_ind.end(),jac.row_indices.begin());
	thrust::copy(d_jac_col_ind.begin(),d_jac_col_ind.end(),jac.column_indices.begin());
	thrust::copy(d_jac_value.begin(),d_jac_value.end(),jac.values.begin());

}

void getDiagonalPrec(cusp::coo_matrix<int, oof_float, cusp_memory> &jact,cusp::array1d<oof_float, cusp_memory> &invCov,cusp::coo_matrix<int, oof_float, cusp_memory> &M){

	M.resize(jact.num_rows,jact.num_rows,jact.num_rows);

	jact.sort_by_row();
	thrust::sequence(M.row_indices.begin(), M.row_indices.end());
	thrust::sequence(M.column_indices.begin(), M.column_indices.end());
	memory_vector<int> outkey(jact.num_rows);

	thrust::equal_to<int> binary_pred;
	thrust::plus<oof_float> binary_op;


	memory_vector<oof_float> v(jact.values.size());
	square_functor<oof_float> sq_op;
	mul_functor<oof_float> mul_op;

	thrust::transform(jact.values.begin(),jact.values.end(),v.begin(),sq_op);

	thrust::transform(
	thrust::make_permutation_iterator(invCov.begin(),jact.column_indices.begin()),
	thrust::make_permutation_iterator(invCov.begin(),jact.column_indices.end()),
	v.begin(),v.begin(),mul_op);

	inv_functor<oof_float> unary_op;
	thrust::reduce_by_key(jact.row_indices.begin(),jact.row_indices.end(),v.begin(),outkey.begin(),M.values.begin(),binary_pred,binary_op);

	thrust::transform(M.values.begin(),M.values.end(),M.values.begin(),unary_op);
}
oof_float getDiagonalMax(cusp::coo_matrix<int, oof_float, cusp_memory> &jact){

	jact.sort_by_row();
	memory_vector<int> outkey(jact.num_rows);
	memory_vector<oof_float> diag(jact.num_rows);

	thrust::equal_to<int> binary_pred;
	thrust::plus<oof_float> binary_op;


	memory_vector<oof_float> v(jact.values.size());
	square_functor<oof_float> sq_op;



	thrust::transform(jact.values.begin(),jact.values.end(),v.begin(),sq_op);
	//squared_plus_functor<oof_float> binary_op;

	thrust::reduce_by_key(jact.row_indices.begin(),jact.row_indices.end(),v.begin(),outkey.begin(),diag.begin(),binary_pred,binary_op);

	oof_float max=thrust::reduce(diag.begin(),diag.end(),0.0,thrust::maximum<oof_float>());
	return max;
}
memory_vector<oof_float> getGaussWeightVector(int row,oof_float mue,oof_float sigma){
	memory_vector<oof_float> x(row);
	memory_vector<oof_float> gaussweight(row);
	thrust::sequence(x.begin(),x.end());
	normal_distribution_functor<oof_float> func(mue,sigma);
	thrust::transform(x.begin(),x.end(),gaussweight.begin(),func);
	thrust::plus<oof_float> plus_func;
	oof_float init=0.0;
	oof_float sum=thrust::reduce(gaussweight.begin(),gaussweight.end(),init,plus_func);
	memory_vector<oof_float> temp(row);
	thrust::fill(temp.begin(),temp.end(),sum);
	thrust::divides<oof_float> divid_func;
	thrust::transform(gaussweight.begin(),gaussweight.end(),temp.begin(),gaussweight.begin(),divid_func);
	return gaussweight;
}



//it returns the global start
int Levmar::registerOptObj(int n){
	int returnInd=nrParaOpt;
	nrParaOpt+=n;
	return returnInd;
}

int Levmar::insertOptObj(void *obj,external_device_set_value_func_t *func,int n,int globalStart){
	for (int i=0;i<n;i++){
		ValueStruct_t v;
		v.m_func=func;
		v.ind=i;
		v.v1=obj;
		d_ValueVec[globalStart+i]=v;
	}
	return 1;
}
int Levmar::registerMeasObj(MeasurementStruct_t *m){
	m->globalStart=nrParaFunc;
	m->startJac=nrParaJac;
	nrParaFunc+=m->nr_row;
	nrParaJac+=m->nr_nonzero_jac;

	return 1;
}
int Levmar::insertMeasObj(MeasurementStruct_t *m){

	m->weight_value=thrust::raw_pointer_cast(&d_weightVec[m->globalStart]);
	m->func_result=thrust::raw_pointer_cast(&d_funcVec[m->globalStart]);
	m->jac_value=thrust::raw_pointer_cast(&d_jac_value[m->startJac]);
	m->jac_row_ind=thrust::raw_pointer_cast(&d_jac_row_ind[m->startJac]);
	m->jac_col_ind=thrust::raw_pointer_cast(&d_jac_col_ind[m->startJac]);
	h_MeasVec.push_back(*m);
	for (int i=0;i<m->nr_row;i++){
		h_inv_covVec[m->globalStart+i]=1.0;
	}

	return 1;
}
int Levmar::insertMeasObj(MeasurementStruct_t *m,std::vector<oof_float> &inv_cov){

	m->weight_value=thrust::raw_pointer_cast(&d_weightVec[m->globalStart]);
	m->func_result=thrust::raw_pointer_cast(&d_funcVec[m->globalStart]);
	m->h_func_result=thrust::raw_pointer_cast(&h_funcVec[m->globalStart]);
	m->jac_value=thrust::raw_pointer_cast(&d_jac_value[m->startJac]);
	m->jac_row_ind=thrust::raw_pointer_cast(&d_jac_row_ind[m->startJac]);
	m->jac_col_ind=thrust::raw_pointer_cast(&d_jac_col_ind[m->startJac]);
	h_MeasVec.push_back(*m);
	for (int i=0;i<m->nr_row;i++){
		if (i<inv_cov.size())
			h_inv_covVec[m->globalStart+i]=inv_cov[i];
		else
			h_inv_covVec[m->globalStart+i]=1.0;
	}

	return 1;
}
int Levmar::initMemory(){

	if (verbose>0)	std::cout<<"nrParameters:"<<nrParaOpt<<"\n";
	if (verbose>0)	std::cout<<"nrParaJac:"<<nrParaJac<<"\n";
	if (verbose>0)	std::cout<<"nrParaFunc:"<<nrParaFunc<<"\n";
	d_weightVec.resize(nrParaFunc);
	d_funcVec.resize(nrParaFunc);
	h_funcVec.resize(nrParaFunc);
	h_inv_covVec.resize(nrParaFunc);
	d_jac_row_ind.resize(nrParaJac);
	d_jac_col_ind.resize(nrParaJac);
	d_jac_value.resize(nrParaJac);
	//h_ValueVec.resize(nrParaOpt);
	d_ValueVec.resize(nrParaOpt);
	return 1;
}
int Levmar::init()
{
#ifdef useHostMemory
	int N=h_MeasVec.size();
	checkCudaErrors(hipHostMalloc((void**)&h_ptr, N*sizeof(MeasurementStruct_t)));
	checkCudaErrors(hipHostGetDevicePointer((void **)&d_ptr, (void *)h_ptr,0));
	d_MeasVec_begin=thrust::device_pointer_cast(d_ptr);
	d_MeasVec_end=	d_MeasVec_begin+N;
	thrust::copy(h_MeasVec.begin(),h_MeasVec.end(),d_MeasVec_begin);
#else
	d_MeasVec.resize(h_MeasVec.size());
	thrust::copy(h_MeasVec.begin(),h_MeasVec.end(),d_MeasVec.begin());
#endif
	return 1;
}


void Levmar::printMemUsage(){
	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	size_t used = total - avail;
	std::cout << "Device memory used: " << used << std::endl;
}



template <class LinearOperator1,
		  class LinearOperator2,
          class Vector,
          class Monitor,
          class Preconditioner>
void cgls(LinearOperator1& At,
		LinearOperator2& A,
        Vector& x,
        Vector& b,
        Vector& invCov,
        Monitor& monitor,
        Preconditioner& M, oof_float mu)
{
   // CUSP_PROFILE_SCOPED();

    typedef typename LinearOperator1::value_type   ValueType;
    typedef typename LinearOperator1::memory_space MemorySpace;
    typedef typename LinearOperator2::value_type   ValueType2;
    typedef typename LinearOperator2::memory_space MemorySpace2;

   // assert(A.num_rows == A.num_cols);        // sanity check

    const size_t N = A.num_cols;
    const size_t N2 = A.num_rows;

    // allocate workspace
    cusp::array1d<ValueType,MemorySpace> y(N);
    cusp::array1d<ValueType,MemorySpace> y1(N2);

    cusp::array1d<ValueType,MemorySpace> z(N);
    cusp::array1d<ValueType,MemorySpace> r(N);
    cusp::array1d<ValueType,MemorySpace> p(N);

    // y <- Ax
    cusp::multiply(A, x, y1);
    blas::xmy(invCov,y1,y1);
    cusp::multiply(At, y1, y);
    blas::axpy(x,y,mu);
    // r <- b - A*x
    blas::axpby(b, y, r, ValueType(1), ValueType(-1));

    // z <- M*r
    cusp::multiply(M, r, z);

    // p <- z
    blas::copy(z, p);

    // rz = <r^H, z>
    ValueType rz = blas::dotc(r, z);

    while (!monitor.finished(r))
    {
        // y <- Ap
        //cusp::multiply(A, p, y);

        cusp::multiply(A, p, y1);
        blas::xmy(invCov,y1,y1);
        cusp::multiply(At, y1, y);
        blas::axpy(p,y,mu);

        // alpha <- <r,z>/<y,p>
        ValueType alpha =  rz / blas::dotc(y, p);
        //std::cout<<"alpha="<<alpha<<"\n";
        // x <- x + alpha * p
        blas::axpy(p, x, alpha);

        // r <- r - alpha * y
        blas::axpy(y, r, -alpha);

        // z <- M*r
        cusp::multiply(M, r, z);

        ValueType rz_old = rz;

        // rz = <r^H, z>
        rz = blas::dotc(r, z);
        //std::cout<<"rz="<<rz<<"\n";
        //std::cout<<"rz_old="<<rz_old<<"\n";

        // beta <- <r_{i+1},r_{i+1}>/<r,r>
        ValueType beta = rz / rz_old;
        if (beta==0.0)
        	break;
        //std::cout<<"beta="<<beta<<"\n";
        // p <- r + beta*p
        blas::axpby(z, p, p, ValueType(1), beta);

        ++monitor;
    }
}

int Levmar::run(){


	timespec ts;
	timespec te;
	clock_gettime(CLOCK_REALTIME, &ts);

	bool found=false;
	double Fx=0.0;
	double Fxnew=0.0;

	k=0;
	residuals.clear();

	//get row and col of jacobian matrix
	row=d_funcVec.size();
	col=d_ValueVec.size();

	//variables for the current state, update state, and function value
	cusp::array1d<oof_float, cusp_memory> x(col);
	cusp::array1d<oof_float, cusp_memory> xnew(col);
	cusp::array1d<oof_float, cusp_memory> f(d_funcVec.size());
	cusp::array1d<oof_float, cusp_memory> w_robust(row);

	cusp::array1d<oof_float, cusp_memory> invCov(row);
	cusp::array1d<oof_float, cusp_memory> invCovRobust(row);
	thrust::copy(h_inv_covVec.begin(),h_inv_covVec.end(),invCov.begin());

	//variables for matrix jac
	cusp::coo_matrix<int, oof_float, cusp_memory> jac(row,col,d_jac_row_ind.size());
	//variables for matrix W for weighting
	//cusp::coo_matrix<int, oof_float, cusp_memory> W(row,row,row);

	//further matrixes needed
	cusp::coo_matrix<int, oof_float, cusp_memory> jact;
	cusp::coo_matrix<int, oof_float, cusp_memory> jactW;
	cusp::coo_matrix<int, oof_float, cusp_memory> jtj;
    cusp::coo_matrix<int, oof_float, cusp_memory> jtj_mueI;
    cusp::coo_matrix<int, oof_float, cusp_memory> mueI(col, col,col);



	cusp::array1d<oof_float, cusp_memory> g(col);
	cusp::array1d<oof_float, cusp_memory> h(col);

    //calculate cost;
	#ifdef useHostMemory
		thrust::for_each(d_MeasVec_begin,d_MeasVec_end,wrapper_functor_with_jac());
		thrust::for_each(d_MeasVec_begin,d_MeasVec_end,robust_functor());
	#else
		thrust::for_each(d_MeasVec.begin(),d_MeasVec.end(),wrapper_functor_with_jac());
		thrust::for_each(d_MeasVec.begin(),d_MeasVec.end(),robust_functor());
	#endif
    thrust::copy(d_weightVec.begin(),d_weightVec.end(),w_robust.begin());
    //copy values from ValueStruct vector to x
	thrust::for_each(d_ValueVec.begin(), d_ValueVec.end(),
			wrapper_functor_set_get(false));


    thrust::transform(d_ValueVec.begin(),d_ValueVec.end(),x.begin(),get_value_functor());

	//copy values from funcVec to f
	thrust::copy(d_funcVec.begin(),d_funcVec.end(),f.begin());

	//create jacobi matrix from vectors
	createMatrix(d_jac_row_ind,d_jac_col_ind,d_jac_value,jac);

	//update matrix struct to have the real size
	jac.resize(row,col,jac.row_indices.size());
	double r=cusp::blas::dot(f,f);

	residuals.push_back(r);

	cusp::transpose(jac, jact);
	//cusp::multiply(jact,W,jactW);

	blas::xmy(invCov,w_robust,invCovRobust);
	blas::xmy(invCovRobust,w_robust,invCovRobust);
	blas::xmy(invCovRobust,f,f);
	//compute g
    cusp::multiply(jact, f, g);

    //-g
    cusp::blas::scal(g,-1.0);

    //compute jact*W*jac
    //cusp::multiply(jact,jac, jtj);

    //get initial mue
    {
        //cusp::array1d<oof_float,cusp_memory> d_diagA;
		//cusp::detail::extract_diagonal(jtj, d_diagA);
		//mue= tau*thrust::reduce(d_diagA.begin(), d_diagA.end(), (oof_float) 0, thrust::maximum<oof_float>());
    	//TODO:: weighted diagonal max nehmen
    	mue= tau*getDiagonalMax(jact);
    	mue=1.0;
	}
    if (verbose>2)	std::cout<<"initial mue:"<<mue<<"\n";
    //check if all values in g (infinity norm) are less than eps1
    max_g = thrust::transform_reduce(g.begin(), g.end(), abs_functor<oof_float>(), 0.0, thrust::maximum<oof_float>());

    //solution found
    if (max_g<eps1)
    	found =true;

    //compute residual same as dot product

    Fx=thrust::transform_reduce(
    		thrust::make_zip_iterator(thrust::make_tuple(f.begin(), invCovRobust.begin())),
    		thrust::make_zip_iterator(thrust::make_tuple(f.end(), invCovRobust.end())),
    		square_weight_functor(), 0.0, thrust::plus<oof_float>()) ;




    //Fx=  thrust::transform_reduce(f.begin(), f.end(), square<oof_float>(), 0.0, thrust::plus<oof_float>()) ;


    if (verbose>0) 	std::cout<<"iteration:"<<k<<" residual:"<<Fx<<"\n";
	clock_gettime(CLOCK_REALTIME, &te);
	if (verbose>3)	std::cout<<"Start Running Loop:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";

    while (!found && (k<kmax)){
    	k++;
    	clock_gettime(CLOCK_REALTIME, &te);
    	if (verbose>3) std::cout<<"Loop:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";

				//create diagonal matrix with mue at the diagonal
				//thrust::sequence(mueI.row_indices.begin(), mueI.row_indices.end());
				//thrust::sequence(mueI.column_indices.begin(), mueI.column_indices.end());
				//thrust::fill(mueI.values.begin(),mueI.values.end(),mue);

    			//(jactWjac+mueI)
    			//cusp::add(jtj,mueI, jtj_mueI);

        if (verbose>3) 	std::cout<<"solve: jact * W * jac    * h  = - g\n";
        thrust::fill(h.begin(),h.end(),0.0);
    //	thrust::copy(g.begin(),g.end(),h.begin());
        cusp::default_monitor<oof_float> monitor(h,cg_it,cg_thresh_rel,cg_thresh_abs);
       // cusp::precond::smoothed_aggregation<int,oof_float,	cusp::device_memory> M(jtj_mueI);
        //cusp::precond::bridson_ainv<oof_float,	cusp::device_memory> M(jtj_mueI);
        //cusp::precond::diagonal<oof_float, cusp_memory> M(jtj_mueI);

        cusp::coo_matrix<int, oof_float, cusp_memory> M;
        getDiagonalPrec(jact,invCovRobust,M);
        // SOLVE jact * W * jac    * h  = - g

        clock_gettime(CLOCK_REALTIME, &te);
    	if (verbose>3) std::cout<<"Solve Normal EQ start:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";
  	    //cusp::krylov::cg(jtj_mueI, h, g,monitor,M);


  	    cgls(jact,jac, h, g,invCovRobust,monitor,M,mue);
  	    clock_gettime(CLOCK_REALTIME, &te);
  	    if (verbose>3) std::cout<<"Solve Normal EQ end:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";
					//cusp::krylov::cg(jtj_mueI, h, g);
					//if (!monitor.converged()){
					//  	    	mue=mue*10.0;
					//  	    	continue;
					//  	    }

    	oof_float hnorm=cusp::blas::nrm2(h);
    	oof_float xnorm=cusp::blas::nrm2(x);

    	if (verbose>4) std::cout<<"hnorm:"<<hnorm<<"\n";
    	//check if change is to small, is so we are finished
    	if (hnorm<eps2*(xnorm+eps2)){
    		found=true;
    		if (verbose>4) std::cout<<"solution found "<<hnorm<<" < " <<eps2*(xnorm+eps2)<<"\n";
    	}else{
    		//xnew = x+h
    		cusp::blas::axpby(x,h,xnew,1.0,1.0);

    		//compute gain
    		cusp::array1d<oof_float, cusp_memory> temp(d_ValueVec.size());
    		//mue*h-g, g is alreade negated
    		cusp::blas::axpby(h,g,temp,mue,1.0);
    		oof_float gainLower=cusp::blas::dot(h,temp);

    		//schreibe werte aus x nach d_value
    		//thrust::copy(xnew.begin(),xnew.end(),d_ValueVec.begin());
    		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(xnew.begin(), d_ValueVec.begin())),
    		thrust::make_zip_iterator(thrust::make_tuple(xnew.end(), d_ValueVec.end())),set_value_functor());

    		if (verbose>4) std::cout<<"copy xnew to original structs\n";
    		//schreibe werte in original structs
    		thrust::for_each(d_ValueVec.begin(),d_ValueVec.end(),wrapper_functor_set_get(true));
    		if (verbose>4) std::cout<<"evaluate function with xnew\n";
    		//evaluate function with xnew
    		clock_gettime(CLOCK_REALTIME, &te);
    		if (verbose>3)
    			std::cout<<"EVAL f and JAC start:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";
			#ifdef useHostMemory
				thrust::for_each(d_MeasVec_begin,d_MeasVec_end,wrapper_functor_with_jac());
				thrust::for_each(d_MeasVec_begin,d_MeasVec_end,robust_functor());
			#else
				thrust::for_each(d_MeasVec.begin(),d_MeasVec.end(),wrapper_functor_with_jac());
				thrust::for_each(d_MeasVec.begin(),d_MeasVec.end(),robust_functor());
			#endif

        	thrust::copy(d_weightVec.begin(),d_weightVec.end(),w_robust.begin());

        	clock_gettime(CLOCK_REALTIME, &te);
        	if (verbose>3) std::cout<<"EVAL F and JAC end:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";
         	//compute Fxnew from evaluation of F with xnew
    		thrust::copy(d_funcVec.begin(),d_funcVec.end(),f.begin());

    		//new residual
    		blas::xmy(invCov,w_robust,invCovRobust);
    		blas::xmy(invCovRobust,w_robust,invCovRobust);
    		blas::xmy(invCovRobust,f,f);
    		Fxnew=thrust::transform_reduce(
    		    		thrust::make_zip_iterator(thrust::make_tuple(f.begin(), invCovRobust.begin())),
    		    		thrust::make_zip_iterator(thrust::make_tuple(f.end(), invCovRobust.end())),
    		    		square_weight_functor(), 0.0, thrust::plus<oof_float>()) ;

//         	Fxnew=  thrust::transform_reduce(f.begin(), f.end(), square<oof_float>(), 0.0, thrust::plus<oof_float>()) ;
         	if (verbose>2) std::cout<<"proposed residual:"<<Fxnew<<"\n";
         	//gain
         	if (gainLower<0)
         		gainLower*=-1.0;
         	oof_float gain=(Fx-Fxnew)/gainLower;

         	if (verbose>4) std::cout<<"gain: "<<gain<<"\n";
         	if (verbose>4) std::cout<<"gainLower: "<<gainLower<<"\n";

         	//if positive gain update x
         	if (gain>0) {
         		//x=xnew
         		thrust::copy(xnew.begin(),xnew.end(),x.begin());
         		//fill jac matrix
         		createMatrix(d_jac_row_ind,d_jac_col_ind,d_jac_value,jac);
         		//copy residuals to f
         		thrust::copy(d_funcVec.begin(),d_funcVec.end(),f.begin());

        		cusp::transpose(jac, jact);
        		 //printMemUsage();

        	//	cusp::multiply(jact,W,jactW);
            	clock_gettime(CLOCK_REALTIME, &te);
            	if (verbose>3) std::cout<<"JacT * Jac start:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";

        		//cusp::multiply(jact,jac, jtj);


        		clock_gettime(CLOCK_REALTIME, &te);
        		if (verbose>3) std::cout<<"JacT * Jac end:"<<(double)(te.tv_sec - ts.tv_sec)<<"sec + "<<(double)(te.tv_nsec - ts.tv_nsec)/1000000 <<"ms\n";
        		blas::xmy(invCov,w_robust,invCovRobust);
        		blas::xmy(invCovRobust,w_robust,invCovRobust);
        		blas::xmy(invCovRobust,f,f);
        	    cusp::multiply(jact, f, g);

        	    cusp::blas::scal(g,-1.0);

        	    //infinity norm of g
        	    max_g = thrust::transform_reduce(g.begin(), g.end(), abs_functor<oof_float>(),(oof_float) 0.0, thrust::maximum<oof_float>());
        	    if (verbose>4) std::cout<<"max g"<<max_g<<"\n";

        	    if (max_g<eps1)
        	    	found =true;
        	    mue*=0.1;
        	    //mue=mue*max(0.33333333334,1.0-pow(2.0*gain-1.0,3));
        	    mue=std::max((oof_float)1e-10,mue);
        	    if (verbose>4) 	std::cout<<"mue:"<<mue<<"\n";

        	    v=2.0;
        	    Fx=thrust::transform_reduce(
        	    		thrust::make_zip_iterator(thrust::make_tuple(f.begin(), invCovRobust.begin())),
        	    		thrust::make_zip_iterator(thrust::make_tuple(f.end(), invCovRobust.end())),
        	    		square_weight_functor(), 0.0, thrust::plus<oof_float>()) ;
             	//Fx=  thrust::transform_reduce(f.begin(), f.end(), square<oof_float>(), 0.0, thrust::plus<oof_float>()) ;

			    if (verbose>1) 	std::cout<<"iteration:"<<k<<" residual:"<<Fx<<"\n";

        	    residuals.push_back(Fx);
		   // if (Fx<0.002)
			//break;
         	}

        	else{
        		residuals.push_back(Fx);
        	    mue=mue*10.0;
        	   // mue=mue*v;
        	    if (verbose>4) 	std::cout<<"mue:"<<mue<<"\n";
        		v=2*v;
        	}

    	}
    }

    if (verbose>0) 	std::cout<<"iteration:"<<k<<" residual:"<<Fx<<"\n";
	//copy solution

	//write value from x to d_value
	//thrust::copy(x.begin(),x.end(),d_ValueVec.begin());
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(x.begin(), d_ValueVec.begin())),
    		thrust::make_zip_iterator(thrust::make_tuple(x.end(), d_ValueVec.end())),set_value_functor());



	//write values to original structs
	thrust::for_each(d_ValueVec.begin(),d_ValueVec.end(),wrapper_functor_set_get(true));
#ifdef useHostMemory
	thrust::for_each(d_MeasVec_begin,d_MeasVec_end,wrapper_functor_with_jac());
#else
	thrust::for_each(d_MeasVec.begin(),d_MeasVec.end(),wrapper_functor_with_jac());
#endif

	thrust::copy(d_funcVec.begin(),d_funcVec.end(),h_funcVec.begin());

	return 1;
}

Levmar::~Levmar() {

}


